#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

// Utilities and system includes
#include <shrUtils.h>
#include <cutil_inline.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;

#ifdef __DEVICE_EMULATION__
const int  NUM_ITERATIONS = 1;
#else
const int  NUM_ITERATIONS = 8192;
#endif


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	printf("[BlackScholes]\n");

  // Start logs
  shrSetLogFileName ("BlackScholes.txt");
  shrLog("%s Starting...\n\n", argv[0]);

    //'h_' prefix - CPU (host) memory space
    float
        //Results calculated by CPU for reference
        *h_CallResultCPU,
        *h_PutResultCPU,
        //CPU copy of GPU results
        *h_CallResultGPU,
        *h_PutResultGPU,
        //CPU instance of input data
        *h_StockPrice,
        *h_OptionStrike,
        *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
        //Results calculated by GPU
        *d_CallResult,
        *d_PutResult,
        //GPU instance of input data
        *d_StockPrice,
        *d_OptionStrike,
        *d_OptionYears;

    double
        delta, ref, sum_delta, sum_ref, max_delta, L1norm, gpuTime;

    unsigned int hTimer;
    int i, devID;
    struct timeval start, end;

/*
FILE* fp_setdevice=fopen("setdevice", "a");
FILE* fp_malloc=fopen("malloc", "a");
FILE* fp_cumalloc=fopen("cumalloc", "a");
FILE* fp_gen=fopen("gen", "a");
FILE* fp_copy1=fopen("copy1", "a");
FILE* fp_copy2=fopen("copy2", "a");
FILE* fp_kernel=fopen("kernel", "a");
FILE* fp_cufree=fopen("cufree", "a");
FILE* fp_free=fopen("free", "a");
*/


/*
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        devID = cutilDeviceInit(argc, argv);
        if (devID < 0) {
           printf("exiting...\n");
           cutilExit(argc, argv);
           exit(0);
	}
    } else {
        hipSetDevice( devID = cutGetMaxGflopsDeviceId() );
    }
*/

    int x = atoi(argv[1]);

//gettimeofday(&start, NULL);
    hipSetDevice(x);
//gettimeofday(&end, NULL);
//fprintf(fp_setdevice, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
//		  - (start.tv_sec * 1000000 + start.tv_usec)));
		
    cutilCheckError( cutCreateTimer(&hTimer) );

    shrLog("Initializing data...\n");
        shrLog("...allocating CPU memory for options.\n");
//gettimeofday(&start, NULL);
        h_CallResultCPU = (float *)malloc(OPT_SZ);
        h_PutResultCPU  = (float *)malloc(OPT_SZ);
        h_CallResultGPU = (float *)malloc(OPT_SZ);
        h_PutResultGPU  = (float *)malloc(OPT_SZ);
        h_StockPrice    = (float *)malloc(OPT_SZ);
        h_OptionStrike  = (float *)malloc(OPT_SZ);
        h_OptionYears   = (float *)malloc(OPT_SZ);
//gettimeofday(&end, NULL);
//fprintf(fp_malloc, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));


        shrLog("...allocating GPU memory for options.\n");
//gettimeofday(&start, NULL);
        cutilSafeCall( hipMalloc((void **)&d_CallResult,   OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_PutResult,    OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_StockPrice,   OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_OptionStrike, OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_OptionYears,  OPT_SZ) );
//gettimeofday(&end, NULL);
//fprintf(fp_cumalloc, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
    //              - (start.tv_sec * 1000000 + start.tv_usec)));


        shrLog("...generating input data in CPU mem.\n");
        srand(5347);
        //Generate options set
//gettimeofday(&start, NULL);
        for(i = 0; i < OPT_N; i++){
            h_CallResultCPU[i] = 0.0f;
            h_PutResultCPU[i]  = -1.0f;
            h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
            h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
            h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
        }
//gettimeofday(&end, NULL);
//fprintf(fp_gen, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));


        shrLog("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing
//gettimeofday(&start, NULL);
        cutilSafeCall( hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice) );
//gettimeofday(&end, NULL);
//fprintf(fp_copy1, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));



    shrLog("Data init done.\n\n");


    shrLog("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
//gettimeofday(&start, NULL);
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        for(i = 0; i < NUM_ITERATIONS; i++){
            BlackScholesGPU<<<480, 128>>>(
                d_CallResult,
                d_PutResult,
                d_StockPrice,
                d_OptionStrike,
                d_OptionYears,
                RISKFREE,
                VOLATILITY,
                OPT_N
            );
            cutilCheckMsg("BlackScholesGPU() execution failed\n");
        }
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
//gettimeofday(&end, NULL);
//fprintf(fp_kernel, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));



        gpuTime = cutGetTimerValue(hTimer) / NUM_ITERATIONS;
        
    //Both call and put is calculated
    shrLog("Options count             : %i     \n", 2 * OPT_N);
    shrLog("BlackScholesGPU() time    : %f msec\n", gpuTime);
    shrLog("Effective memory bandwidth: %f GB/s\n", ((double)(5 * OPT_N * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    shrLog("Gigaoptions per second    : %f     \n\n", ((double)(2 * OPT_N) * 1E-9) / (gpuTime * 1E-3));

    shrLogEx(LOGBOTH | MASTER, 0, "BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u options, NumDevsUsed = %u, Workgroup = %u\n", 
           (((double)(2.0 * OPT_N) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime*1e-3, (2 * OPT_N), 1, 128);

    shrLog("\nReading back GPU results...\n");
        //Read back GPU results to compare them to CPU results
//gettimeofday(&start, NULL);
        cutilSafeCall( hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy(h_PutResultGPU,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost) );
//gettimeofday(&end, NULL);
//fprintf(fp_copy2, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));


/*
    shrLog("Checking the results...\n");
        shrLog("...running CPU calculations.\n\n");
        //Calculate options values on CPU
        BlackScholesCPU(
            h_CallResultCPU,
            h_PutResultCPU,
            h_StockPrice,
            h_OptionStrike,
            h_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );

        shrLog("Comparing the results...\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        max_delta = 0;
        for(i = 0; i < OPT_N; i++){
            ref   = h_CallResultCPU[i];
            delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);
            if(delta > max_delta) max_delta = delta;
            sum_delta += delta;
            sum_ref   += fabs(ref);
        }
        L1norm = sum_delta / sum_ref;
        shrLog("L1 norm: %E\n", L1norm);
        shrLog("Max absolute error: %E\n\n", max_delta);
*/


    shrLog("Shutting down...\n");
        shrLog("...releasing GPU memory.\n");
//gettimeofday(&start, NULL);
        cutilSafeCall( hipFree(d_OptionYears)  );
        cutilSafeCall( hipFree(d_OptionStrike) );
        cutilSafeCall( hipFree(d_StockPrice)  );
        cutilSafeCall( hipFree(d_PutResult)    );
        cutilSafeCall( hipFree(d_CallResult)   );
//gettimeofday(&end, NULL);
//fprintf(fp_cufree, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));


        shrLog("...releasing CPU memory.\n");
//gettimeofday(&start, NULL);
        free(h_OptionYears);
        free(h_OptionStrike);
        free(h_StockPrice);
        free(h_PutResultGPU);
        free(h_CallResultGPU);
        free(h_PutResultCPU);
        free(h_CallResultCPU);
        cutilCheckError( cutDeleteTimer(hTimer) );
//gettimeofday(&end, NULL);
//fprintf(fp_free, "%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
  //                - (start.tv_sec * 1000000 + start.tv_usec)));


	shrLog("Shutdown done.\n");

	printf("\n[BlackScholes] - Test Summary\n");
	shrLog((L1norm < 1e-6) ? "PASSED\n\n" : "FAILED\n\n");


    hipDeviceReset();

    shrEXIT(argc, (const char**)argv);
}
